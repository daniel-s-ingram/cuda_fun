#include <cuda_fun/GridInterface.hpp>
#include <cuda_fun/Vector.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include <iostream>

#define cudaCheckError(code) { cudaAssert((code), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
    if (code == hipSuccess) 
    {
        return;
    }

    printf("%s in file %s on line %d\n\n", hipGetErrorString(code), file, line);
    exit(1);
}

namespace cuda_fun
{

template<typename GridCellType>
GridInterface<GridCellType>::GridInterface(const std::uint32_t rows, const std::uint32_t cols, GridCellType* const h_grid) :
    m_rows{rows},
    m_cols{cols},
    m_size{m_rows*m_cols*sizeof(GridCellType)},
    m_h_grid{nullptr},
    m_d_current_grid{nullptr},
    m_d_next_grid{nullptr}
{
    cudaCheckError(hipMalloc((void**)&m_d_current_grid, m_size));
    cudaCheckError(hipMalloc((void**)&m_d_next_grid, m_size));

    if (h_grid != nullptr)
    {
        m_h_grid = h_grid;
        cudaCheckError(hipMemcpy(m_d_current_grid, m_h_grid, m_size, hipMemcpyHostToDevice));
    }
}

template<typename GridCellType>
GridInterface<GridCellType>::~GridInterface()
{
    cudaCheckError(hipFree(m_d_current_grid));
    cudaCheckError(hipFree(m_d_next_grid));
}

template class GridInterface<std::uint8_t>;
template class GridInterface<float>;
template class GridInterface<Vec3f>;

}