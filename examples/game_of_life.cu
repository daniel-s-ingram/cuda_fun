#include "hip/hip_runtime.h"
#include <cuda_fun/GridInterface.hpp>
#include <cuda_fun/GridVisualizer.hpp>

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cstdint>
#include <ctime>
#include <iostream>
#include <memory>
#include <random>

#define cudaCheckError(code) { cudaAssert((code), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
    if (code == hipSuccess) 
    {
        return;
    }

    printf("%s in file %s on line %d\n\n", hipGetErrorString(code), file, line);
    exit(1);
}

namespace cuda_fun
{

__global__ void doGpuGol(const std::uint8_t* const current_grid, std::uint8_t* const next_grid, const int N)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int i = by*blockDim.y + ty;
    const int j = bx*blockDim.x + tx;

    const std::size_t up    = (i > 0) ? (i - 1) : (N - 1);
    const std::size_t down  = (i < (N - 1)) ? (i + 1) : 0;
    const std::size_t left  = (j > 0) ? (j - 1) : (N - 1);
    const std::size_t right = (j < (N - 1)) ? (j + 1) : 0;

    int num_live_neighbors{0};
    num_live_neighbors += (!current_grid[up*N + left]) ? 0 : 1;
    num_live_neighbors += (!current_grid[up*N + j]) ? 0 : 1;
    num_live_neighbors += (!current_grid[up*N + right]) ? 0 : 1;
    num_live_neighbors += (!current_grid[i*N + right]) ? 0 : 1;
    num_live_neighbors += (!current_grid[down*N + right]) ? 0 : 1;
    num_live_neighbors += (!current_grid[down*N + j]) ? 0 : 1;
    num_live_neighbors += (!current_grid[down*N + left]) ? 0 : 1;
    num_live_neighbors += (!current_grid[i*N + left]) ? 0 : 1;

    if (current_grid[i*N + j])
    {
        //printf("Alive! %d\n", num_live_neighbors);
        if (num_live_neighbors < 2U || num_live_neighbors > 3U)
        {
            next_grid[i*N + j] = 0;
        }
        else
        {
            next_grid[i*N + j] = current_grid[i*N + j];
        }
        
    }
    else
    {
        //printf("Dead! %d\n", num_live_neighbors);
        if (num_live_neighbors == 3U)
        {
            next_grid[i*N + j] = 255;
        }
        else
        {
            next_grid[i*N + j] = current_grid[i*N + j];
        }
    }
}

class GameOfLife : public GridInterface<std::uint8_t>
{
public:
    GameOfLife(const std::size_t rows, const std::size_t cols, std::uint8_t* const h_grid = nullptr) : 
        GridInterface<std::uint8_t>(rows, cols, h_grid)
    {}

    void update() override
    {
        hipDeviceSynchronize();

        const dim3 block_dim{32, 32, 1};
        const dim3 grid_dim{m_rows / block_dim.x, m_cols / block_dim.y, 1};
        doGpuGol<<<grid_dim, block_dim>>>(m_d_current_grid, m_d_next_grid, m_rows);
        cudaCheckError(hipPeekAtLastError());

        hipDeviceSynchronize();

        std::swap(m_d_current_grid, m_d_next_grid);

        cudaCheckError(hipMemcpy(m_h_grid, m_d_current_grid, m_size, hipMemcpyDeviceToHost));
    }
};

}

template<typename T, typename... Args>
std::unique_ptr<T> make_unique(Args&&... args)
{
    return std::unique_ptr<T>(new T(std::forward<Args>(args)...));
}

void populateGrid(std::uint8_t* const grid, const int N)
{
    std::srand(std::time(nullptr));
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            grid[i*N + j] = (0.4 > std::rand()/static_cast<double>(RAND_MAX)) ? 255 : 0;
        }
    }
}

int main()
{
    constexpr std::size_t rows{8196};
    constexpr std::size_t cols{rows};

    std::uint8_t* const h_grid = new std::uint8_t[rows*cols];
    populateGrid(h_grid, rows);

    using namespace cuda_fun;
    GridVisualizer grid_visualizer{rows, cols};
    std::unique_ptr<GridInterface<std::uint8_t>> game_of_life = make_unique<GameOfLife>(rows, cols, h_grid);

    grid_visualizer.run(std::move(game_of_life));

    return 0;
}