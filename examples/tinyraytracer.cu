#include "hip/hip_runtime.h"
#include <cuda_fun/GridInterface.hpp>
#include <cuda_fun/GridVisualizer.hpp>
#include <cuda_fun/Vector.hpp>
#include <cuda_fun/Sphere.hpp>

#include <cmath>
#include <iostream>
#include <limits>
#include <vector>

#define cudaCheckError(code) { cudaAssert((code), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
    if (code == hipSuccess) 
    {
        return;
    }

    hipDeviceReset();
    printf("%s in file %s on line %d\n\n", hipGetErrorString(code), file, line);
    exit(1);
}

namespace cuda_fun
{

void populateGrid(Vec3f* const grid, const int N)
{
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            grid[i*N + j] = Vec3f{j/float(N), i/float(N), 0};
        }
    }
}

struct Material
{
    __host__ __device__ Material(const Vec3f& color) : 
        diffuse_color{color}
    {}

    __host__ __device__ Material() : diffuse_color{}
    {}

    Vec3f diffuse_color;
};

struct Light
{
    Light(const Vec3f& p, const float i) :
        position{p},
        intensity{i}
    {}

    Vec3f position;
    float intensity;
};

struct Sphere
{
    Vec3f center;
    Material material;
    float radius;

    Sphere(const Vec3f& c, const Material& m, const float r) : 
        center{c},
        material{m},
        radius{r}
    {}

    __host__ __device__ bool ray_intersect(const Vec3f& orig, const Vec3f& dir, float& t0) const 
    {
        const Vec3f L = center - orig;
        const float tca = L*dir;
        const float d2 = L*L - tca*tca;
        if (d2 > radius*radius)
        {
            return false;
        }

        const float thc = std::sqrt(radius*radius - d2);
        const float t1 = tca + thc;

        t0 = tca - thc;
        if (t0 < 0)
        {
            t0 = t1;
        }

        if (t0 < 0)
        {
            return false;
        }

        return true;
    }
};

__host__ __device__ bool scene_intersect(const Vec3f& orig, const Vec3f& dir, const Sphere* const spheres, const int num_spheres, Vec3f& hit, Vec3f& N, Material& material) {
    float spheres_dist = std::numeric_limits<float>::max();
    for (int i = 0; i < num_spheres; ++i) {
        float dist_i;
        if (spheres[i].ray_intersect(orig, dir, dist_i) && dist_i < spheres_dist) {
            spheres_dist = dist_i;
            hit = orig + dir*dist_i;
            N = (hit - spheres[i].center).normalized();
            material = spheres[i].material;
        }
    }
    
    return spheres_dist<1000;
}

__host__ __device__ Vec3f cast_ray(const Vec3f &orig, const Vec3f &dir, const Sphere* const spheres, const int num_spheres, const Light* const lights, const int num_lights) 
{
    Vec3f point, N;
    Material material;

    float sphere_dist = std::numeric_limits<float>::max();
    if (!scene_intersect(orig, dir, spheres, num_spheres, point, N, material))
    {
        return Vec3f(0.2, 0.7, 0.8);
    }

    float diffuse_light_intensity = 1.0;
    for (std::size_t i = 0; i < num_lights; ++i) {
        // const Vec3f light_dir = (lights[i].position - point).normalized();
        // diffuse_light_intensity += lights[i].intensity * std::max(0.f, light_dir*N);
    }
    
    return material.diffuse_color * diffuse_light_intensity;
}

__global__ void render(Vec3f* const current_grid, const Sphere* const spheres, const int N, const int num_spheres, const Light* const lights, const int num_lights)
{
    constexpr int fov = M_PI/2.;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int i = by*blockDim.y + ty;
    const int j = bx*blockDim.x + tx;

    const float x =  (2*(i + 0.5)/(float)N - 1)*std::tan(fov/2.)*N/(float)N;
    const float y = -(2*(j + 0.5)/(float)N - 1)*std::tan(fov/2.);
    const Vec3f dir = Vec3f{x, y, -1}.normalized();

    const Vec3f origin{0, 0, 1};
    current_grid[i*N+j] = cast_ray(origin, dir, spheres, num_spheres, lights, num_spheres);
}

__global__ void moveSpheres(Sphere* const spheres, const int num_spheres)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= num_spheres)
    {
        return;
    }

    spheres[i].center += Vec3f{0.01, 0.01, 0.0};
}

class TinyRayTracer : public GridInterface<Vec3f>
{
public:
    TinyRayTracer(const std::size_t rows, const std::size_t cols, Vec3f* const h_grid, const std::vector<Sphere>& spheres, const std::vector<Light>& lights) : 
        GridInterface<Vec3f>(rows, cols, h_grid)
    {    
        m_num_spheres = spheres.size();
        m_spheres_size = m_num_spheres*sizeof(Sphere);

        m_num_lights = lights.size();
        m_lights_size = m_num_lights*sizeof(Light);

        cudaCheckError(hipMalloc((void**)&m_d_spheres, m_spheres_size));
        cudaCheckError(hipMalloc((void**)&m_d_lights, m_lights_size));
        
        cudaCheckError(hipMemcpy(m_d_spheres, spheres.data(), m_spheres_size, hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(m_d_lights, lights.data(), m_lights_size, hipMemcpyHostToDevice));

        std::cout << "spheres: " << m_d_spheres << " " << m_d_spheres + m_spheres_size << std::endl;
        std::cout << "lights " << m_d_lights << " " << m_d_lights + m_lights_size << std::endl;
    }

    void update() override
    {
        const dim3 block_dim{32, 32, 1};
        const dim3 grid_dim{m_rows / block_dim.x, m_cols / block_dim.y, 1};
        render<<<grid_dim, block_dim>>>(m_d_current_grid, m_d_spheres, m_rows, m_num_spheres, m_d_lights, m_num_lights);
        cudaCheckError(hipPeekAtLastError());

        moveSpheres<<<1, m_num_spheres>>>(m_d_spheres, m_num_spheres);

        //std::swap(m_d_current_grid, m_d_next_grid);

        // todo: figure out how to draw directly from GPU memory to avoid this copy
        //std::cout << m_h_grid << std::endl;
        cudaCheckError(hipMemcpy(m_h_grid, m_d_current_grid, m_size, hipMemcpyDeviceToHost));
    }

    ~TinyRayTracer()
    {
        cudaCheckError(hipFree(m_d_spheres));
        cudaCheckError(hipFree(m_d_lights));
    }

private:
    Sphere* m_d_spheres;
    Light* m_d_lights;
    std::size_t m_num_spheres;
    std::size_t m_spheres_size;
    std::size_t m_num_lights;
    std::size_t m_lights_size;
};

}

int main()
{
    using namespace cuda_fun;

    constexpr std::size_t rows{1024};
    constexpr std::size_t cols{rows};

    Vec3f* const h_grid = new Vec3f[rows*cols];
    populateGrid(h_grid, rows);

    std::vector<Sphere> spheres;
    spheres.emplace_back(Vec3f{-3, -3, -10}, Material{Vec3f{0.5, 0, 0}}, 1);
    spheres.emplace_back(Vec3f{-2, -8, -20}, Material{Vec3f{0.2, 0.7, 0}}, 4);
    spheres.emplace_back(Vec3f{-5, -5, -13}, Material{Vec3f{0.2, 0.2, 0.4}}, 2);

    std::vector<Light> lights;
    //lights.emplace_back(Vec3f{-20, 20, 20}, 0.5);

    GridVisualizer grid_visualizer{rows, cols};
    std::unique_ptr<GridInterface<Vec3f>> tiny_ray_tracer = std::make_unique<TinyRayTracer>(rows, cols, h_grid, spheres, lights);

    grid_visualizer.run(std::move(tiny_ray_tracer));

    return 0;
}